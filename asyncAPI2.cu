#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

/**
 * 需求：需要把若干个一维数组传给核函数
 * 实现方法：在gpu生成一个一维的指针数组，每个元素指向一个普通一维数组。
 * 把该指针数组的地址传递给核函数。
 * 其实该指针数组充当二维数组的角色。
 */

__global__ void
testKernel(float ** pointerArray)
{
	printf("(%d,%d):%f\n",threadIdx.y,threadIdx.x,pointerArray[threadIdx.y][threadIdx.x]);
}


int main()
{

	int rows = 2;
	int cols = 2;


	float** host_2d = new float*[rows];
	float** dev_2d;
	for (int k = 0; k < rows; k++)
	{
		float* host_1d = new float[cols];
		//---------下面两个是测试数据，看看核函数printf输出的是否与这个一致------------------------------
		host_1d[0] = 99;
		host_1d[1] = 100;

		float* dev_1d ;
		hipMalloc((void**)&dev_1d,sizeof(float)*cols);//该指针指向的是一个float数组
		hipMemcpy(dev_1d,host_1d,sizeof(float)*cols,hipMemcpyHostToDevice);

		host_2d[k] = dev_1d;
	}
	hipMalloc((void**)&dev_2d,sizeof(float*)*rows);
	hipMemcpy(dev_2d,host_2d,sizeof(float*)*rows,hipMemcpyHostToDevice);

	dim3 threads(rows,cols);
	testKernel<<<1,threads>>>(dev_2d);

	hipDeviceSynchronize();

	//记得释放内存啊，养成好习惯。这里省去这部分代码




return 0;
}
