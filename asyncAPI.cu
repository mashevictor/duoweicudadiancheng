
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixs_1D_multiplication(int *matrix_a_dev,int *matrix_b_dev,int *matrix_c_dev,int row,int col)//记住这里的row和col直接对应global里面的数值，不能有误
{
    int j = threadIdx.x+blockIdx.x * blockDim.x;
    int i = threadIdx.y+blockIdx.y * blockDim.y;

    if(i< row &&j < row)
    {
        for(int k = 0; k < col; k++)
        {
            matrix_c_dev[row *i + j] += matrix_a_dev[i* col + k] * matrix_b_dev[row*k + j];
        }
    }
}

int main()
{
    int row = 4;
    int col = 5;
    int *matrix_a_host;
    int *matrix_b_host;
    int *matrix_c_host;

    matrix_a_host = (int *)malloc(row*col*sizeof(int));
    matrix_b_host = (int *)malloc(row*col*sizeof(int));
    matrix_c_host = (int *)malloc(row*row*sizeof(int));

    for(int i = 0; i<row; i++)
    {
        for(int j = 0; j < col; j++)
        {
            matrix_a_host[i*col +j] = i+j;
        }
    }

    printf("\n-------------Matrix a-----------------\n");
    for(int i = 0; i < row*col; i++)
    {
        printf("%d ",*(matrix_a_host + i));
        if(i%col==col-1) printf("\n");//每输出3个换行。
    }


    for(int i = 0; i<col; i++)
    {
        for(int j = 0; j < row; j++)
        {
            matrix_b_host[i*row +j] = i+j;
        }
    }



    // ------------------GPU--------------------------
    int *matrix_a_dev;
    int *matrix_b_dev;
    int *matrix_c_dev;

    hipMalloc((void**) &matrix_a_dev, row*col*sizeof(int));
    hipMalloc((void**) &matrix_b_dev, row*col*sizeof(int));
    hipMalloc((void**) &matrix_c_dev, row*row*sizeof(int));

    hipMemcpy(matrix_a_dev, matrix_a_host, row*col*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(matrix_b_dev, matrix_b_host, row*col*sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(1, 2, 1);
    dim3 dimBlock(4, 5, 1);//记住这里的row和col直接对应global里面的数值，不能有误
    matrixs_1D_multiplication<<<dimGrid, dimBlock>>>(matrix_a_dev, matrix_b_dev, matrix_c_dev, row,col);

    hipMemcpy(matrix_c_host, matrix_c_dev, row*row*sizeof(int), hipMemcpyDeviceToHost);

    printf("\n-------------Matrix c-----------------\n");
    for(int i = 0; i < row*row; i++)
    {
        printf("%d ",*(matrix_c_host + i));
        if(i%row==row-1) printf("\n");//每输出4个换行。
    }

    free(matrix_a_host);
    free(matrix_b_host);
    free(matrix_c_host);
    hipFree(matrix_a_dev);
    hipFree(matrix_b_dev);
    hipFree(matrix_c_dev);

    return 1;
}
